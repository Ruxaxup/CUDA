#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : ProyectoCUDA.cu
 Author      : MTI
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals

 Cmd: nvcc `pkg-config --cflags opencv` ProyectoCUDA.cu `pkg-config --libs opencv` -o proyecto
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>


//Librerias de OpenCV
#include <opencv2/opencv.hpp>
#include <opencv2/highgui/highgui.hpp>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

#define IMG_WIDTH 256
#define IMG_HEIGHT 256
#define THREADS 100 //x 10
#define BLOCKS 66

using namespace std;
using namespace cv;

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void comparamela(unsigned char *d_MA,unsigned char *d_MB,unsigned char *d_MC) {
	int id = blockIdx.x * blockDim.x * blockDim.y
				+ threadIdx.y * blockDim.x + threadIdx.x;

	if(id < IMG_WIDTH*IMG_WIDTH)
		d_MC[id] = d_MA[id] - d_MB[id];
	//d_MC[id] = 200;
}

int main(void)
{
	unsigned char matrizA[IMG_WIDTH * IMG_HEIGHT];
	unsigned char matrizB[IMG_WIDTH * IMG_HEIGHT];
	unsigned char matrizC[IMG_WIDTH * IMG_HEIGHT];

	unsigned char *d_MA, *d_MB, *d_MC;

	//Cargar imageness
	char* name1 = "78.jpg";
	char* name2 = "83.jpg";
	Mat imageA;
	Mat imageB;
	
	imageA = imread(name1,1);
	imageB = imread(name2,1);
	namedWindow("Display Image", WINDOW_AUTOSIZE );
	imshow("Display Image", imageA);
	//////////////////////
	//Copiar imagenes a arreglos
	Vec3b intensityA,intensityB;
	for(int i=0; i<IMG_WIDTH; i++){
		for(int j=0; j<IMG_WIDTH; j++){
		    intensityA = imageA.at<Vec3b>(i, j);
		    intensityB = imageB.at<Vec3b>(i, j);
		    matrizA[i*IMG_WIDTH+j]=(unsigned char)intensityA.val[2];
		    matrizB[i*IMG_WIDTH+j]=(unsigned char)intensityB.val[2];
		    matrizC[i*IMG_WIDTH+j] = 0;
		}
	}
	
	hipMalloc((void**)&d_MA,sizeof(char)*IMG_HEIGHT*IMG_WIDTH);
	hipMalloc((void**)&d_MB,sizeof(char)*IMG_HEIGHT*IMG_WIDTH);
	hipMalloc((void**)&d_MC,sizeof(char)*IMG_HEIGHT*IMG_WIDTH);

	hipMemcpy(d_MA,matrizA,sizeof(char)*IMG_WIDTH*IMG_HEIGHT,hipMemcpyHostToDevice);
	hipMemcpy(d_MB,matrizB,sizeof(char)*IMG_WIDTH*IMG_HEIGHT,hipMemcpyHostToDevice);

	dim3 bloque(BLOCKS);
	dim3 hilos(10,THREADS);

	comparamela<<<bloque,hilos>>>(d_MA,d_MB,d_MC);



	hipMemcpy(matrizC,d_MC,sizeof(char)*IMG_HEIGHT*IMG_WIDTH,hipMemcpyDeviceToHost);
	int iteraciones = 0;
	for(int i=0; i<IMG_WIDTH && iteraciones < 40; i++){
		for(int j=0; j<IMG_WIDTH; j++){
			if(matrizC[i*IMG_WIDTH+j] != 0){
				iteraciones++;
				printf("A %i ",matrizA[i*IMG_WIDTH+j]);
				printf("B %i ",matrizB[i*IMG_WIDTH+j]);
			   	printf("C %i -- ",matrizC[i*IMG_WIDTH+j]);
			}					    
		}
		printf("\n");
	}

	Mat imagedif;
	imagedif = imageB;
	Vec3b intensityC;
	//Se recrea la imagen a partir del arreglo c
	    for(int i=0; i<IMG_WIDTH; i++){
			for(int j=0; j<IMG_WIDTH; j++){
			    intensityC.val[0] = matrizC[i*IMG_WIDTH+j];
			    intensityC.val[1] = matrizC[i*IMG_WIDTH+j];
			    intensityC.val[2] = matrizC[i*IMG_WIDTH+j];
			    imagedif.at<Vec3b>(i, j)=intensityC;
			}
	    }
	    namedWindow( "diferencia de im", CV_WINDOW_NORMAL );
	    imshow( "diferencia de im", imagedif);
	waitKey(0);

	/* Free memory */
	hipFree(d_MA);
	hipFree(d_MB);
	hipFree(d_MC);
	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

